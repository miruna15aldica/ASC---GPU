#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
// Functia pentru cautarea valorii lui nounce
__global__ void findNonce(BYTE *difficulty, BYTE *content, uint64_t *nonce) {
	// Daca nounce-ul a fost gasit deja
	if(*nonce != 0)
		return;
    BYTE copy_content[BLOCK_SIZE];
	char nonce_str[NONCE_SIZE];
    BYTE block_hash[SHA256_HASH_SIZE];
	// Identificatorul unic al threadului
    uint64_t id_thread = blockIdx.x * blockDim.x + threadIdx.x;
	// Copiem continutul blocului
    d_strcpy((char*)copy_content, (const char*)content);
    intToString(id_thread, nonce_str);
	// Adaugam nounce-ul la sfarsit
    d_strcpy((char*)copy_content + d_strlen((const char*)copy_content), nonce_str);
    // Functia de hash
	apply_sha256(copy_content, d_strlen((const char*)copy_content), block_hash, 1);
    // Daca hash-ul are dificultatea potrivita
	if (compare_hashes(block_hash, difficulty) <= 0) {
		// schimbam nounce-ul cu valoarea pe care am gasit-o
		atomicExch((unsigned long long *)nonce,(unsigned long long)id_thread);
    }
}

// Alocam memorie pe dispozitivul CUDA
void mallocsCuda(BYTE **device_difficulty, BYTE **device_content, uint64_t **device_nonce) {
    hipMalloc((void **)device_difficulty, SHA256_HASH_SIZE);
    hipMalloc((void **)device_content, BLOCK_SIZE);
    hipMalloc((void **)device_nonce, sizeof(uint64_t));
}

// Eliberam memorie pe dispozitivul CUDA
void freeCuda(BYTE *device_difficulty, BYTE *device_content, uint64_t *device_nonce) {
    hipFree(device_difficulty);
    hipFree(device_content);
    hipFree(device_nonce);
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint64_t nonce = 0;
	size_t current_length;

	// Rezultat nounce
	char nonce_result[NONCE_SIZE];

	// Variabile pentru stocarea datelor pe CUDA
    BYTE *device_difficulty;
    BYTE *device_content;
    uint64_t *device_nonce;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	// Alocare memorie
    mallocsCuda(&device_difficulty, &device_content, &device_nonce);

	// Transferam datele pe dispozitivul CUDA
    hipMemcpy(device_difficulty, DIFFICULTY, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_nonce, &nonce, sizeof(uint64_t), hipMemcpyHostToDevice);

	// Cautam nounce-ul
	findNonce<<< 1 + MAX_NONCE / 256, 256>>>(device_difficulty, device_content, device_nonce);
    hipMemcpy(&nonce, device_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
    // Convertim nonce-ul in sir de caractere + actualizam blocu;
	snprintf((char *)block_content + strlen((char*) block_content), BLOCK_SIZE - strlen((char*) block_content), "%llu", nonce);
	int len = strlen((const char *)block_content + strlen((char*) block_content))+ strlen((char*) block_content);
	// Hash actualizat	
	apply_sha256(block_content, len, block_hash, 1);

    freeCuda(device_difficulty, device_content, device_nonce);
	float seconds = stopTiming(&start, &stop);
	printResult(block_hash, nonce, seconds);

	return 0;
}
